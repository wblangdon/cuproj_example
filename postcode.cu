#include "hip/hip_runtime.h"
/*WBL 19 Dec 2025 Based on my example.cu r1.10
  https://github.com/rapidsai/cuspatial/tree/branch-24.12/cpp/cuproj#example

Modifications:
WBL 31 Dec 2024 revert r1.15, cf https://github.com/rapidsai/cuspatial/issues/1510#issuecomment-2566227767, speed up with 4e6 coords 494
#   read data created by make_tests.bat r1.5 from postcode.in
WBL 30 Dec 2024 add calling run_proj_test
WBL 29 Dec 2024 try 4 inputs, add run_proj_test
*/

/* UK post codes in oops/h3/open_postcode_geo.csv.gz
 * NB some post do not have lat and long, so take care of defaults or clean
 */

using namespace std;

#include <cuproj_test/convert_coordinates.hpp>
#include <cuproj_test/coordinate_generator.cuh>
#include <cuspatial_test/vector_equality.hpp>

#include <cuproj/error.hpp>
#include <cuproj/projection_factories.cuh>
#include <cuproj/vec_2d.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <proj.h>

#include <fstream>
#include <iostream>
//https://stackoverflow.com/questions/997946/how-to-get-current-time-and-date-in-c
#include <chrono>
#include <ctime>

template <typename T>
using coordinate = typename cuspatial::vec_2d<T>;

//#include <iomanip>

//run_proj_test() etc from cuspatial-branch-24.12 wgs_to_utm_test.cu
// run a test using the proj library for comparison
void run_proj_test(thrust::host_vector<PJ_COORD>& coords,
                   char const* epsg_src,
                   char const* epsg_dst)
{
  PJ_CONTEXT* C = proj_context_create();
  PJ* P         = proj_create_crs_to_crs(C, epsg_src, epsg_dst, nullptr);
  proj_trans_array(P, PJ_FWD, coords.size(), coords.data());

  proj_destroy(P);
  proj_context_destroy(C);
}

int main() {                            
  cout << "Start $Revision: 1.23 $\n";

  using T = float;

  //https://forums.developer.nvidia.com/t/using-thrust-copy-to-copy-from-a-file-to-a-device-vector/25956/3
  ifstream ifile("postcode.in");
  if (!ifile) {throw std::runtime_error("ifile postcode.in error");}
  const int num = 2589831;
  thrust::host_vector<T> H(num*2);
  istream_iterator<T> beg(ifile), end;
  thrust::copy(beg, end, H.begin());
  assert(std::abs(H[2*num-1] - (-1.307578)) < 1e-5);//check last value
  //ofstream ofile("postcode.out");
  //thrust::copy(H.begin(),H.end(),
  //	       ostream_iterator<T>(ofile, "\n"));
  ifile.close();
  //ofile.close();
  //return 0;
  
  //cuproj::vec_2d<T> sydney{-33.858700, 151.214000};  // Sydney, NSW, Australia
  thrust::host_vector<cuspatial::vec_2d<T>> input(num);
  for(int i = 0; i < input.size(); i++){
    input[i] = {H[i*2],H[i*2+1]}; //{sydney.x+float(i/400000.0),sydney.y};
  }
  assert(std::abs(input[num-1].x - (59.887104)) < 1e-5);//check last x value
  assert(std::abs(input[num-1].y - (-1.307578)) < 1e-5);//check last y value

  thrust::host_vector<coordinate<T>> h_input(input.begin(), input.end());
  thrust::host_vector<PJ_COORD> pj_input{input.size()};
  cuproj_test::convert_coordinates(h_input, pj_input);
  thrust::host_vector<PJ_COORD> pj_expected(pj_input);

  char const* epsg_src = "EPSG:4326";
  char const* epsg_dst = "EPSG:32756"; //"utm_epsg.c_str();
/*for(int i = 0; i <input.size(); i++) {
    cout << "input: " << std::setprecision(20) <<
      pj_expected[i].v[0] << "," <<
      pj_expected[i].v[1] << "," <<
      pj_expected[i].v[2] << "," <<
      pj_expected[i].v[3] << endl;
  }*/
  std::chrono::duration<double> pj_elapsed_seconds;
  {cout << "run_proj_test "<<flush;
  auto start = std::chrono::system_clock::now();
  run_proj_test(pj_expected, epsg_src, epsg_dst);
  auto end = std::chrono::system_clock::now();
  pj_elapsed_seconds = end-start;
  cout << "  done run_proj_test " << pj_elapsed_seconds.count() << endl;}
  
/*for(int i = 0; i <input.size(); i++) {
    cout << "pj_expected: " << std::setprecision(20) <<
      pj_expected[i].v[0] << "," <<
      pj_expected[i].v[1] << "," <<
      pj_expected[i].v[2] << "," <<
      pj_expected[i].v[3] << endl;
  }*/
  
  // Make a projection to convert WGS84 (lat, lon) coordinates to UTM zone 56S (x, y) coordinates
  auto* proj = cuproj::make_projection<cuproj::vec_2d<T>>("EPSG:4326", "EPSG:32756");

  /*https://docs.nvidia.com/cuda/archive/9.0/pdf/Thrust_Quick_Start_Guide.pdf
  // H has storage for 4 integers
  thrust::host_vector<T> H(4);
  // initialize individual elements
  H[0] = 14;
  H[1] = 20;
  H[2] = 38;
  H[3] = 46;

  // H.size() returns the size of vector H
  std::cout << "H has size " << H.size() << std::endl;
  // print contents of H
  for(int i = 0; i < H.size(); i++)
    std::cout << "H[" << i << "] = " << H[i] << std::endl;
  // resize H
  H.resize(2);
  std::cout << "H now has size " << H.size() << std::endl;
  // Copy host_vector H to device_vector D
  thrust::device_vector<int> D = H;
  // elements of D can be modified
  D[0] = 99;
  D[1] = 88;
  // print contents of D
  for(int i = 0; i < D.size(); i++)
    std::cout << "D[" << i << "] = " << D[i] << std::endl;
  // H and D are automatically deleted when the function returns
  return 0;
  */
  thrust::host_vector<cuproj::vec_2d<T>> h_in(input.size());
  for(int i = 0; i <h_in.size(); i++) {
    h_in[i] = {input[i].x,input[i].y};
  }
  thrust::device_vector<cuproj::vec_2d<T>> d_in = h_in; //transfer to GPU
  thrust::device_vector<cuproj::vec_2d<T>> d_out(d_in.size());

  // Convert the coordinates. Works the same with a vector of many coordinates.
  {cout << "proj->transform "<<flush;
  auto start = std::chrono::system_clock::now();
  proj->transform(d_in.begin(), d_in.end(), d_out.begin(), cuproj::direction::FORWARD);
  auto end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end-start;
  cout << "done proj->transform " << elapsed_seconds.count() << endl;
  cout << "Speedup "<<pj_elapsed_seconds.count()/elapsed_seconds.count()<<endl;}

  //Check answer (based on wgs_to_utm_test.cu run_cuproj_test)
/*for(int i = 0; i < d_out.size(); i++) {
    cout << "pj_expected: " << std::setprecision(20) <<
      pj_expected[i].v[0] << "," <<
      pj_expected[i].v[1] << "," <<
      pj_expected[i].v[2] << "," <<
      pj_expected[i].v[3] << endl;
    cout << "Device:     "  << std::setprecision(20) << d_out[i] << endl;
  }*/


  // We can expect nanometer accuracy with double precision. The precision ratio of
  // double to single precision is 2^53 / 2^24 == 2^29 ~= 10^9, then we should
  // expect meter (10^9 nanometer) accuracy with single precision.
  //T tolerance = std::is_same_v<T, double> ? T{1e-9} : T{1.0};
  // We can expect nanometer accuracy with double precision. The precision ratio of
  // double to single precision is 2^53 / 2^24 == 2^29 ~= 10^9, then we should
  // expect meter (10^9 nanometer) accuracy with single precision.
  // For large arrays seem to need to relax the tolerance a bit to match PROJ results.
  // 1um for double and 10m for float seems like reasonable accuracy while not allowing excessive
  // variance from PROJ results.
  T tolerance = std::is_same_v<T, double> ? T{1e-6} : T{10};
  cout << "tolerance: " << tolerance << endl;
  thrust::device_vector<cuspatial::vec_2d<T>> expected(pj_input.size());
  for(int i = 0; i < pj_input.size(); i++){
    expected[i] = {float(pj_expected[i].v[0]),float(pj_expected[i].v[1])};
  }
  thrust::device_vector<cuspatial::vec_2d<T>> c_out(d_out.size());
  for(int i = 0; i < d_out.size(); i++){
    cuproj::vec_2d<T> c = d_out[i];
    //cout << (d_out[i]).x <<endl; https://stackoverflow.com/questions/8638839/thrust-vector-of-type-uint2-has-no-member-x-compiler-error
    //cout << c <<endl;
    //cout << " " << c.x << "," << c.y << endl;
    c_out[i] = {c.x,c.y}; //{1,2}; //{float(d_out[i].x),float(d_out[i].y)};
  }
  {cout << "CUSPATIAL_EXPECT_VECTORS_EQUIVALENT "<<flush;
  auto start = std::chrono::system_clock::now();
  CUSPATIAL_EXPECT_VECTORS_EQUIVALENT(expected, c_out, tolerance);
  auto end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end-start;
  cout << "done CUSPATIAL_EXPECT_VECTORS_EQUIVALENT " << elapsed_seconds.count() << endl;}
  
  cout << "end main().\n";
}
